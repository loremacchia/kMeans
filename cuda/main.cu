#include "hip/hip_runtime.h"
#include "rapidcsv.h"
#include <float.h>
#include <hip/hip_runtime.h>


//Function to make atomicAdd usable for double
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

//Function declarations
double* getDataset();
int getDataLength();
double* getFarCentroids(double *points, int pointsLength, int dimensions);
__global__ void assignClusterReduction(int length, int dimensions, int numBlocks, double *points, double *centroids, double *blocksCentroids, int *blocksPointPerCluster);

//Project parameters
const int k = 3;
const int dimensions = 2;
const int threadPerBlock = 1024;


int main(int argc, char const *argv[]) {
    double *points = getDataset(); // Getting the dataset from the file dataset.csv
    int dataLength = getDataLength(); // Length of the dataset
    double *centroids = getFarCentroids(points, dataLength, dimensions); // Cluster initialization
    int numBlocks = (dataLength + threadPerBlock - 1)/threadPerBlock; // Number of blocks of threads to be created in computation

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    //Allocate device memory to work with global memory 
    double *points_dev; // Device memory copy of dataset points
    hipMalloc(&points_dev, dataLength*dimensions*sizeof(double));
    hipMemcpy(points_dev, points, dataLength*dimensions*sizeof(double), hipMemcpyHostToDevice);

    double *centroids_dev; // Device memory copy of old centroids
    hipMalloc(&centroids_dev, k*dimensions*sizeof(double));
    
    double *blocksCentroids; // Device memory copy of new centroids (in update)
    hipMalloc(&blocksCentroids, k*dimensions*numBlocks*sizeof(double)); 

    int *blocksPointPerCluster; // Device memory copy of partial number of points in each cluster (in update)
    hipMalloc(&blocksPointPerCluster, k*numBlocks*sizeof(int));


    double distanceFromOld = 0; // Variable to chek in the stopping condition. It is the distance of the new set of centroids wrt the old one
    // Representation of a cluster i: centroids[i*dimensions:(i+1)*dimensions-1], pointsInCluster[i], newCentroids[i*dimensions:(i+1)*dimensions-1]
    int pointsInCluster[k]; // Number of points in a cluster i
    double *newCentroids = new double[k*dimensions]; // Temp values of the evaluated new centroids for each cluster
    int iter = 0; // Counter to verify how many loop iterations are done by the algorithm

    // Loop to calculate the final clusters
    do {
        // Copy the newCentroids in device memory as old centroids and init the updated values of new centroid and points in cluster
        hipMemcpy(centroids_dev, centroids, k*dimensions*sizeof(double), hipMemcpyHostToDevice);
        hipMemset(blocksCentroids, 0, k*dimensions*numBlocks*sizeof(double)); 
        hipMemset(blocksPointPerCluster, 0, k*numBlocks*sizeof(int)); 
        
        // Function that calls numBlocks*threadPerBlock threads and evaluates the new cluster values
        assignClusterReduction<<<numBlocks, threadPerBlock>>>(dataLength, dimensions, numBlocks, points_dev, centroids_dev, blocksCentroids, blocksPointPerCluster);
        
        // Error checking and thread synchronization
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
        hipDeviceSynchronize();
        
        // Copying the returned values of centroids and points in clusters from device memory to host memory
        hipMemcpy(newCentroids, blocksCentroids, k*dimensions*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(pointsInCluster, blocksPointPerCluster, k*sizeof(int), hipMemcpyDeviceToHost);
    
        // Setting the correct values of newCentroids and updating distanceFromOld and centroids to the actual values
        distanceFromOld = 0;
        for (int j = 0; j < k; j++) {
            for (int x = 0; x < dimensions; x++) {
                newCentroids[j*dimensions + x] /= pointsInCluster[j];
                distanceFromOld += fabs(newCentroids[j*dimensions + x] - centroids[j*dimensions + x]);
                centroids[j*dimensions + x] = newCentroids[j*dimensions + x];
            }
        }
        iter++;
    } while (distanceFromOld > 0.001); // Check stopping condition
    // Deallocating device memory
    hipFree(points_dev);
    hipFree(centroids_dev);
    hipFree(blocksCentroids);
    hipFree(blocksPointPerCluster);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float outerTime;
    hipEventElapsedTime( &outerTime, start, stop ); // Computation time
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Write the computational time into a CSV file
    std::ofstream myfile;
    myfile.open ("./cuda/cuda.csv", std::ios::app);
    myfile << dataLength;
    myfile << "," << outerTime;
    myfile << "\n";
    myfile.close();

    printf("\n\ncuda: %f\n\n\n",outerTime);
    return 0;
}

// Function to evaluate the actual point assignment to the correct cluster and to aggregate all the results with a reduction
// newCentroids and pointsInCluster are updated in block's shared memory, then it will do a reduction
__global__ void assignClusterReduction(int length, int dim, int numBlocks,double *points, double *centroids, double *blocksCentroids, int *blocksPointPerCluster){
    // Block copy in shared memory of the block partial results of newCentroids and pointsInCluster
    __shared__ double newCentroids[dimensions*k]; 
    __shared__ int pointsInCluster[k];
    // Init shared memory variables using the first dimension*k threads
    if(threadIdx.x < dimensions*k) {
        newCentroids[threadIdx.x] = 0;
    }
    if(threadIdx.x < k) {
        pointsInCluster[threadIdx.x] = 0;
    }
    __syncthreads(); 
        
    // If the thread has an associated point it evaluates the cluster to be assigned and it does an atomic add to the block's cluster partial results
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < length) {
        double dist = FLT_MAX; // Updated distance from point to the nearest Cluster
        int clustId = -1; // Id of the nearest Cluster
        for (int j = 0; j < k; j++) {
            double newDist = 0; //Distance from each Cluster
            for (int x = 0; x < dimensions; x++) {
                newDist += fabs(points[idx*dimensions + x] - centroids[j*dimensions + x]);
            }
            if(newDist < dist) {
                dist = newDist;
                clustId = j;
            }            
        }        
        for (int x = 0; x < dimensions; x++) {
            atomicAdd(&(newCentroids[clustId*dimensions + x]), points[idx*dimensions + x]);
        }
        atomicAdd(&(pointsInCluster[clustId]),1);
    }
    __syncthreads();

    // Finally the firsts dimension*k threads of each block add the local block cluster values to the global ones with a reduction
    if(threadIdx.x < dimensions*k) {
        atomicAdd(&(blocksCentroids[threadIdx.x]), newCentroids[threadIdx.x]);
    }
    if(threadIdx.x < k) {
        atomicAdd(&(blocksPointPerCluster[threadIdx.x]), pointsInCluster[threadIdx.x]);
    }
}


// Getting data length
int getDataLength(){
    rapidcsv::Document doc("./dataset.csv", rapidcsv::LabelParams(-1, -1));
    return int(doc.GetRowCount()) - k;
}

// Getting the dataset from the CSV file. The last k values are the correct centroids
double* getDataset() {
    rapidcsv::Document doc("./dataset.csv", rapidcsv::LabelParams(-1, -1));
    const int rows = int(doc.GetRowCount()) - k;
    const int dimensions = doc.GetColumnCount() - 1;
    double *points = new double[rows*dimensions];
    for(int i = 0; i < rows; i++) {
        std::vector<std::string> row = doc.GetRow<std::string>(i);  
        double *array = new double[dimensions];
        int index = 0;
        for(auto element : row) {
            if(index != dimensions) {
                points[i*dimensions + index] = std::atof(element.c_str());
            }
            index++;
        }
    }
    return points;
}

// Centroids initialization function
// The centroids are: a random point from the set (for us the first) and the k-1 furthest points of the set
double* getFarCentroids(double *points, int pointsLength, int dimensions) {
    // Init set of clusters picking a point from the set and the k - 1 points further wrt the chosen point.
    // Those will be the firts clustroids
    double reference[dimensions]; // The first available point is chosen as reference
    for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
        reference[tmpIdx] = points[tmpIdx];
    }
    
    double *distances = new double[k-1]; 
    double *realPoints = new double[k*dimensions]; // Array containing the furthest points wrt reference
    for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
        realPoints[(k-1)*dimensions + tmpIdx] = reference[tmpIdx];
    }
    
    int maxSize = k - 1; // Maximum size of the array and relative indexes
    // Get the k - 1 points
    for(int i = 0; i < pointsLength; i++){
        double dist = 0;
        for (int x = 0; x < dimensions; x++) {
            dist += fabs(points[i*dimensions + x] - reference[x]);
        }
        if(dist > distances[maxSize - 1]) { // If the distance is higher than the last element of the array
            int index = 0;
            while (dist < distances[index] && index < maxSize) { // Find the right place to put in the point
                index++;
            }
            for (int j = maxSize - 1; j > index; j--) {
                distances[j] = distances[j - 1];
                for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
                    realPoints[j*dimensions + tmpIdx] = realPoints[(j - 1)*dimensions + tmpIdx];
                }
            }
            distances[index] = dist;
            for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
                realPoints[index*dimensions + tmpIdx] = points[i*dimensions + tmpIdx];
            }
        }
    }
    return realPoints;
}