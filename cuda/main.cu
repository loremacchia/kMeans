#include "hip/hip_runtime.h"
#include "rapidcsv.h"
#include <math.h> 
#include <hip/hip_runtime.h>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

double* getDataset(int* lenght, int* dim);
double* getFarCentroids(double *points, int pointsLength, int dimensions);
__global__ void assignCluster(int length, int dimensions, double *points, double *centroids, int *pointsInCluster, double *newCentroids);

const int k = 3;

int main(int argc, char const *argv[]) {
    int dataLength;
    int dimensions;
    double *points = getDataset(&dataLength, &dimensions);
    double *centroids = getFarCentroids(points, dataLength, dimensions);

    for (int i = 0; i < dataLength; i++) {
        for (int j = 0; j < dimensions; j++) {
            printf("%f ", points[i*dimensions + j]);
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < dimensions; j++) {
            printf("%f ", centroids[i*dimensions + j]);
        }
        printf("\n");
    }
    
    double *points_dev;
    hipMalloc(&points_dev, dataLength*dimensions*sizeof(double));
    hipMemcpy(points_dev, points, dataLength*dimensions*sizeof(double), hipMemcpyHostToDevice);

    double *centroids_dev;
    hipMalloc(&centroids_dev, k*dimensions*sizeof(double));
    

    double *newCentroids_dev;
    hipMalloc(&newCentroids_dev, k*dimensions*sizeof(double));

    int *pointsInCluster_dev;
    hipMalloc(&pointsInCluster_dev, k*sizeof(int));

    double distanceFromOld = 0;
    int *pointsInCluster = new int[k]; 
    double *newCentroids = new double[k*dimensions];

    do {
        hipMemcpy(centroids_dev, centroids, k*dimensions*sizeof(double), hipMemcpyHostToDevice);
        hipMemset(newCentroids_dev, 0, k*dimensions*sizeof(double));
        hipMemset(pointsInCluster_dev, 0, k*sizeof(int));
        
        assignCluster<<<(dataLength +127)/128, 128>>>(dataLength, dimensions, points_dev, centroids_dev, pointsInCluster_dev, newCentroids_dev);
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
        hipDeviceSynchronize();

        hipMemcpy(newCentroids, newCentroids_dev, k*dimensions*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(pointsInCluster, pointsInCluster_dev, k*sizeof(int), hipMemcpyDeviceToHost);

        distanceFromOld = 0;
        for (int j = 0; j < k; j++) {
            printf("%d ",pointsInCluster[j]);
            for (int x = 0; x < dimensions; x++) {
                printf("%f ",newCentroids[j*dimensions + x] );
                newCentroids[j*dimensions + x] /= pointsInCluster[j];
            }
            printf("\n");
        }
        for (int j = 0; j < k; j++) {
            for (int x = 0; x < dimensions; x++) {
                distanceFromOld += fabs(newCentroids[j*dimensions + x] - centroids[j*dimensions + x]);
            }
        }
        for (int j = 0; j < k; j++) {
            for (int x = 0; x < dimensions; x++) {
                centroids[j*dimensions + x] = newCentroids[j*dimensions + x];
            }
        }
        printf("%f\n",distanceFromOld);
    } while (distanceFromOld > 0.0001);
    
    hipFree(points_dev);
    hipFree(centroids_dev);
    hipFree(newCentroids_dev);
    hipFree(pointsInCluster_dev);
    return 0;
}

//FILE 2
//TODO newCentroids e pointsInCluster devono essere scritti, quindi potrebbe essere buono fare variabili shared
//centroids deve essere solo letto e mai modificato, quindi metterlo in constant memory?
//points sono troppi da tenere tutti in memoria della gpu, dove e come si spezzettano? Metterli in LOCAL memory o REGISTER solamente 1 per ogni thread (o metterli nella shared)
__global__ void assignCluster(int length, int dimensions, double *points, double *centroids, int *pointsInCluster, double *newCentroids){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < length) {
        double dist = 100; // Updated distance from point to the nearest Cluster. Init with a big value. TODO check if it is enough
        int clustId = -1; // Id of the nearest Cluster

        for (int j = 0; j < k; j++) {
            double newDist = 0; //Distance from each Cluster
            for (int x = 0; x < dimensions; x++) {
                newDist += fabsf(points[idx*dimensions + x] - centroids[j*dimensions + x]);
            }
            if(newDist < dist) {
                dist = newDist;
                clustId = j;
            }
            printf("%f, %d, %d\n",newDist,idx,j);
        }
        printf("%d - %d\n",idx, clustId);
        __syncthreads();
        
        //TODO atomic
        for (int x = 0; x < dimensions; x++) {
            printf("%d -- %d -- %d -- %f\n", idx, clustId, x, newCentroids[clustId*dimensions + x]);
            atomicAdd(&(newCentroids[clustId*dimensions + x]), points[idx*dimensions + x]);
            printf("%d -- %d -- %d -- %f\n", idx, clustId, x, newCentroids[clustId*dimensions + x]);
        }
        printf("%d -- %d -- %d\n", idx, clustId, pointsInCluster[clustId]);
        atomicAdd(&(pointsInCluster[clustId]),1);
        printf("%d -- %d -- %d\n", idx, clustId, pointsInCluster[clustId]);
    }
}



double* getDataset(int* lenght, int* dim) {
    rapidcsv::Document doc("./dataset.csv", rapidcsv::LabelParams(-1, -1));
    const int rows = int(doc.GetRowCount()) - k;
    const int dimensions = doc.GetColumnCount() - 1;
    *lenght = rows;
    *dim = dimensions;
    printf("%d\n",rows);
    double *points = new double[rows*dimensions];
    for(int i = 0; i < rows; i++) {
        std::vector<std::string> row = doc.GetRow<std::string>(i);  
        double *array = new double[dimensions];
        int index = 0;
        for(auto element : row) {
            if(index != dimensions) {
                // std::cout << element << std::endl;
                points[i*dimensions + index] = std::atof(element.c_str());
            }
            index++;
        }
    }
    return points;
}

double* getFarCentroids(double *points, int pointsLength, int dimensions) {
    // Init set of clusters picking a point from the set and the k - 1 points further wrt the chosen point.
    // Those will be the firts clustroids
    double reference[dimensions]; // The first available point is chosen as reference
    for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
        reference[tmpIdx] = points[tmpIdx];
    }
    
    double *distances = new double[k-1]; 
    double *realPoints = new double[k*dimensions]; // Array containing the further points wrt reference
    for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
        realPoints[(k-1)*dimensions + tmpIdx] = reference[tmpIdx];
    }
    
    int maxSize = k - 1; // Maximum size of the array and relative indexes
    // Get the k - 1 points
    for(int i = 0; i < pointsLength; i++){
        double dist = 0;
        for (int x = 0; x < dimensions; x++) {
            dist += fabs(points[i*dimensions + x] - reference[x]);
        }
        if(dist > distances[maxSize - 1]) { // If the distance is higher than the last element of the array
            int index = 0;
            while (dist < distances[index] && index < maxSize) { // Find the right place to put in the point
                index++;
            }
            for (int j = maxSize - 1; j > index; j--) {
                distances[j] = distances[j - 1];
                for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
                    realPoints[j*dimensions + tmpIdx] = realPoints[(j - 1)*dimensions + tmpIdx];
                }
            }
            distances[index] = dist;
            for (int tmpIdx = 0; tmpIdx < dimensions; tmpIdx++) {
                realPoints[index*dimensions + tmpIdx] = points[i*dimensions + tmpIdx];
            }
        }
    }
    return realPoints;
}